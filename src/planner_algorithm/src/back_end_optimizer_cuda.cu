#include "hip/hip_runtime.h"
#include <planner_algorithm/back_end_optimizer.h>
#include <planner_algorithm/lbfgs.hpp>

#include <hip/hip_runtime.h>

__device__ bool grad_cost_p(void* ptrObj,
                          const Eigen::Vector3d& p,
                          Eigen::Vector3d& gradp,
                          double& costp) {
  TrajOpt& obj = *(TrajOpt*)ptrObj;
  double sdf_value = 0 ;
  double truncation = 1.0;
  double travel_cost = 0;
  Eigen::Vector3d gp(Eigen::Vector3d::Zero());
  Eigen::Vector3d gtr(Eigen::Vector3d::Zero());
  costp = 0;
  gradp = Eigen::Vector3d::Zero();
  sdf_value = (obj.environment->occupancy_map) -> getDistWithGradTrilinear(p, gp);
  // travel cost
  travel_cost = (obj.environment->travelcost_map) -> getSDFValue(p);
  gtr         = -(obj.environment->travelcost_map) -> getSDFGrad(p);

  if ( sdf_value <=  truncation && sdf_value != 0 )
  {    
    costp +=  obj.rhoP * pow (truncation - sdf_value, 3);
    gradp +=  obj.rhoP * 3 * pow(truncation - sdf_value, 2) * (-gp);
  }
  if( travel_cost > 0)
  {
    costp +=  obj.rhoP * pow( travel_cost,3 );
    gradp +=  obj.rhoP * 3 * pow( travel_cost,2 ) * gtr;
  }
  return true;
}

__device__ bool grad_cost_v(void* ptrObj,
                          const Eigen::Vector3d& v,
                          Eigen::Vector3d& gradv,
                          double& costv) {
  
  TrajOpt& obj = *(TrajOpt*)ptrObj;
  double vpen  = v.squaredNorm() - (obj.vmax) * (obj.vmax);
  if (vpen > 0) {
    gradv = obj.rhoV * 6 * vpen * vpen * v;
    costv = obj.rhoV * vpen * vpen * vpen;
    return true;
  }
  return false;
}

__device__ bool grad_cost_a(void* ptrObj,
                          const Eigen::Vector3d& a,
                          Eigen::Vector3d& grada,
                          double& costa) {

  TrajOpt& obj = *(TrajOpt*)ptrObj;
  grada = Eigen::Vector3d::Zero();
  costa = 0;
  double apen  = a.squaredNorm() - (obj.amax) * (obj.amax);

  if (apen > 0) {
    grada += (obj.rhoA) * 6 * apen * apen * a;
    costa += (obj.rhoA) * apen * apen * apen;
    return true;
  }
 
  return false;
}

__global__ void addCost(void* ptrObj, double* cost, int n)
{
    TrajOpt& obj = *(TrajOpt*)ptrObj;
    Eigen::Vector3d pos, vel, acc, jer;
    Eigen::Vector3d grad_tmp, grad_tmp_p, grad_tmp_v ;
    double cost_tmp, cost_tmp_p, cost_tmp_v;
    Eigen::Matrix<double, 6, 1> beta0, beta1, beta2, beta3;
    double s1, s2, s3, s4, s5;
    double step, alpha;
    Eigen::Matrix<double, 6, 3> gradViolaPc, gradViolaVc, gradViolaAc;
    double gradViolaPt, gradViolaVt, gradViolaAt;
    double omg;
    int innerLoop;

    // 获取全局索引
    int i = blockIdx.x;
    int j = threadIdx.x;
    ///////////////////////
    innerLoop = obj.K + 1;
    const auto& c = obj.jerkOpt.b.block<6, 3>(i * 6, 0);
    s1 = 0.0;
    step = obj.jerkOpt.T1(i) / obj.K;
    for(int l = 0; l < i; l++){ s1 += obj.jerkOpt.T1(l); }
    s1 += step * j;

    s2 = s1 * s1;
    s3 = s2 * s1;
    s4 = s2 * s2;
    s5 = s4 * s1;
    beta0 << 1.0, s1, s2, s3, s4, s5;
    beta1 << 0.0, 1.0, 2.0 * s1, 3.0 * s2, 4.0 * s3, 5.0 * s4;
    beta2 << 0.0, 0.0, 2.0, 6.0 * s1, 12.0 * s2, 20.0 * s3;
    beta3 << 0.0, 0.0, 0.0, 6.0, 24.0 * s1, 60.0 * s2;
    alpha = 1.0 / obj.K * j;
    pos = c.transpose() * beta0;
    vel = c.transpose() * beta1;
    acc = c.transpose() * beta2;
    jer = c.transpose() * beta3;

    omg = (j == 0 || j == innerLoop - 1) ? 0.5 : 1.0;

    if (grad_cost_p(ptrObj,pos, grad_tmp, cost_tmp)) {
        
      gradViolaPc = beta0 * grad_tmp.transpose();
      gradViolaPt = alpha * grad_tmp.dot(vel);
      obj.jerkOpt.gdC.block<6, 3>(i * 6, 0) += omg * step * gradViolaPc;
      obj.jerkOpt.gdT(i) += omg * (cost_tmp / obj.K + step * gradViolaPt);
      (*cost) += omg * step * cost_tmp;
    }
    if (grad_cost_v(ptrObj,vel, grad_tmp, cost_tmp)) {
      gradViolaVc = beta1 * grad_tmp.transpose();
      gradViolaVt = alpha * grad_tmp.dot(acc);
      obj.jerkOpt.gdC.block<6, 3>(i * 6, 0) += omg * step * gradViolaVc;
      obj.jerkOpt.gdT(i) += omg * (cost_tmp / obj.K + step * gradViolaVt);
      (*cost) += omg * step * cost_tmp;
    }
    if (grad_cost_a(ptrObj,acc, grad_tmp, cost_tmp)) {
      gradViolaAc = beta2 * grad_tmp.transpose();
      gradViolaAt = alpha * grad_tmp.dot(jer);
      obj.jerkOpt.gdC.block<6, 3>(i * 6, 0) += omg * step * gradViolaAc;
      obj.jerkOpt.gdT(i) += omg * (cost_tmp / obj.K + step * gradViolaAt);
      (*cost) += omg * step * cost_tmp;
    }
}


__global__ void TrajOpt::addTimeIntPenalty(double& cost) {

    int nBytes = 1 * sizeof(double);
    // 申请host内存
    double *cost_host;
      cost_host  = (double*)malloc(nBytes);
    (*cost_host) = 0;

    // 申请device内存
    double *cost_device;
    hipMalloc((void**)&cost_device, nBytes);

    // 将host数据拷贝到device
    hipMemcpy((void*)cost_device, (void*)cost_host, nBytes, hipMemcpyHostToDevice);

    // 定义kernel的执行配置
    dim1 blockSize( (this->N) );
    dim1 gridSize(  (this->K) + 1 );
    // 执行kernel
    addCost << < gridSize, blockSize >> >( this, cost_device, this->N);

    // 将device得到的结果拷贝到host
    hipMemcpy((void*)cost_host, (void*)cost_device, nBytes, hipMemcpyDeviceToHost);

    // 检查执行结果
    cout << "cost: " << (*cost_host) << endl;
    hipFree(cost_device);
    free(cost_host);

    cost = *(cost_host);
}




static double rhoP_tmp_;

// SECTION  variables transformation and gradient transmission
__host__ static double expC2(double t) {
  return t > 0.0 ? ((0.5 * t + 1.0) * t + 1.0)
                 : 1.0 / ((0.5 * t - 1.0) * t + 1.0);
}

__host__ static double logC2(double T) {
  return T > 1.0 ? (sqrt(2.0 * T - 1.0) - 1.0) : (1.0 - sqrt(2.0 / T - 1.0));
}

__host__ static inline double gdT2t(double t) {
  if (t > 0) {
    return t + 1.0;
  } else {
    double denSqrt = (0.5 * t - 1.0) * t + 1.0;
    return (1.0 - t) / (denSqrt * denSqrt);
  }
}

__host__ static void forwardT(const double& t, Eigen::Ref<Eigen::VectorXd> vecT) {
  vecT.setConstant(expC2(t));
}

__host__ static void addLayerTGrad(const double& t,
                          const Eigen::Ref<const Eigen::VectorXd>& gradT,
                          double& gradt) {
  gradt = gradT.sum() * gdT2t(t);
}

// !SECTION variables transformation and gradient transmission

// SECTION object function
__host__ static inline double objectiveFunc(void* ptrObj,
                                   const double* x,
                                   double* grad,
                                   const int n) {
  TrajOpt& obj = *(TrajOpt*)ptrObj;
  //const double& t = x[0];
  //double& gradt = grad[0];
  VectorXd VT(obj.N);
  Eigen::Map<const Eigen::MatrixXd> T( x , 1, (obj.dim_t) );
  Eigen::Map<const Eigen::MatrixXd> P(x + (obj.dim_t), 3, (obj.dim_p) );
  Eigen::Map<Eigen::MatrixXd> gradT(grad , 1, (obj.dim_t) );
  Eigen::Map<Eigen::MatrixXd> gradP(grad + (obj.dim_t), 3, (obj.dim_p) );

  //VT = T.row(0);
  double t = T(0,0);
  //VT = VectorXd::Ones( (obj.N) ) * T(0,0);
  forwardT(t, VT);
  (obj.jerkOpt).generate(P, (obj.finalS), VT);

  double cost = (obj.jerkOpt).getTrajJerkCost();
  (obj.jerkOpt).calGrads_CT();
  obj.addTimeIntPenalty(cost);

  (obj.jerkOpt).calGrads_PT();
  (obj.jerkOpt).gdT.array() += (obj.rhoT);

  cost += (obj.rhoT) * VT.sum();

  addLayerTGrad(t , (obj.jerkOpt).gdT, gradT(0,0));
  //gradT = (obj.jerkOpt).gdT.transpose();
  gradP = (obj.jerkOpt).gdP;

  return cost;
}


// !SECTION object function
__host__ static inline int earlyExit(void* ptrObj,
                            const double* x,
                            const double* grad,
                            const double fx,
                            const double xnorm,
                            const double gnorm,
                            const double step,
                            int n,
                            int k,
                            int ls) {
  TrajOpt& obj = *(TrajOpt*)ptrObj;
  //if (obj.pause_debug_) {
  if (false) {
    
    VectorXd VT(obj.N);
    Eigen::Map<const Eigen::MatrixXd> T( x , 1, (obj.dim_t) );
    Eigen::Map<const Eigen::MatrixXd> P( x + (obj.dim_t) , 3, (obj.dim_p) );

    //VT = VectorXd::Ones( (obj.N) ) * T(0,0);
    
    forwardT(T(0,0), VT);
    obj.jerkOpt.generate(P, obj.finalS, VT);
    auto traj = obj.jerkOpt.getTraj();
    obj.drawDebug(traj, P);


    // NOTE pause
    std::this_thread::sleep_for(std::chrono::milliseconds(10));
  }
  return k > 1e3;
}

__host__ bool TrajOpt::generate_traj(const Eigen::MatrixXd& initState,
                            const Eigen::MatrixXd& finalState,
                            const std::vector<Eigen::Vector3d>& Q,
                            const int N,
                            Trajectory& traj,
                            bool keep_result) {

  this->N = N;
  
  
  //this->dim_t = N;
  this->dim_t = 1;
  this->dim_p = N - 1;

  this->x = new double[ (this->dim_t) + 3 * (this->dim_p)];

  VectorXd VT(N);
  Eigen::Map<Eigen::MatrixXd> T( (this->x) , 1, (this->dim_t) );
  Eigen::Map<Eigen::MatrixXd> P( (this->x) + (this->dim_t) , 3, (this->dim_p) );

  // NOTE set boundary conditions
  (this->initS)  = initState;
  (this->finalS) = finalState;
  double tempNorm = (this->initS).col(1).norm(); // v0
  (this->initS).col(1) *= tempNorm > (this->vmax) ? ((this->vmax) / tempNorm) : 1.0;
  tempNorm = (this->initS).col(2).norm(); //a0
  (this->initS).col(2) *= tempNorm > (this->amax) ? ((this->amax) / tempNorm) : 1.0;

  // set initial guess
  /* 
  double len0 = (initState.col(0)  - Q[0]).norm();
  double lenf = (finalState.col(0) - Q[N-1]).norm();
  T(0,0)   = len0 / (this->vmax);
  T(0,N-1) = lenf / (this->vmax);
  for (int i = 1; i < N - 1; i++)
  {
    T(0,i) =  (Q[i]  - Q[i-1]).norm() / (this->vmax);
  }
  */
  double len = 0.0;
  len += (initState.col(0)  - Q[0]).norm(); 
  len += (finalState.col(0)  - Q[N-1]).norm();
  for (int i = 1; i < N - 1; i++)
  {
    len +=  (Q[i]  - Q[i-1]).norm() ;
  }
  double T0 = len / N / (this->vmax);
  //T(0,0) = T0;
  T(0,0) = logC2(T0);
  //t = logC2(T0);
 
  for (int i = 0; i < N - 1; ++i) {
    P.col(i) = Q[i];
  }
  (this->jerkOpt).reset(initState, N);
  // NOTE optimization
  lbfgs::lbfgs_parameter_t lbfgs_params;
  lbfgs::lbfgs_load_default_parameters(&lbfgs_params);
  lbfgs_params.mem_size = 128;
  lbfgs_params.past = 3;
  lbfgs_params.g_epsilon = 1e-32;
  lbfgs_params.min_step = 1e-32;
  lbfgs_params.delta = 1e-5;
  lbfgs_params.line_search_type = 0;
  double minObjectiveXY , minObjectiveZ;

  rhoP_tmp_ = (this->rhoP);


  auto opt_ret1 = lbfgs::lbfgs_optimize((this->dim_t) + 3 * (this->dim_p), 
                                       this->x, 
                                       &minObjectiveXY,
                                       &objectiveFunc, nullptr,
                                       &earlyExit, this, &lbfgs_params);

  std::cout << "\033[32m"
            << "ret: " << opt_ret1 << "\033[0m" << std::endl;
  if (this->pause_debug) {
    std::this_thread::sleep_for(std::chrono::milliseconds(2000));
  }
  if (opt_ret1 < 0) {
    if( !keep_result )
    {
      delete[] this->x;
    }
    return false;
  }

  forwardT(T(0,0), VT);
  //VT = VectorXd::Ones(N) * T(0,0);
  (this->jerkOpt).generate(P, finalState, VT);
  traj = (this->jerkOpt).getTraj();
  if( !keep_result )
  {
    delete[] this->x;
  }
  
  return true;
}


__host__ void TrajOpt::drawDebug(Trajectory end_path , Eigen::Map<const Eigen::MatrixXd> P)
{
  int id = 0;
  visualization_msgs::Marker sphere, line_strip;
  sphere.header.frame_id = line_strip.header.frame_id = "world";
  sphere.header.stamp = line_strip.header.stamp = ros::Time::now();
  sphere.type = visualization_msgs::Marker::SPHERE_LIST;
  line_strip.type = visualization_msgs::Marker::LINE_STRIP;
  sphere.action = line_strip.action = visualization_msgs::Marker::ADD;
  sphere.id = id;
  line_strip.id = id + 1000;
  id++;

  sphere.pose.orientation.w = line_strip.pose.orientation.w = 1.0;
  sphere.color.r = 1;
  sphere.color.g = 0.5;
  sphere.color.b = 0.5;
  sphere.color.a = 1;
  line_strip.color.r = 1;
  line_strip.color.g = 0;
  line_strip.color.b = 1;
  line_strip.color.a = 1;
  sphere.scale.x = 0.1;
  sphere.scale.y = 0.1;
  sphere.scale.z = 0.1;
  line_strip.scale.x = 0.05 / 2;
  geometry_msgs::Point pt;

  double dur = end_path.getDurations().sum();
  for (double i = 0; i < dur - 1e-4; i+=0.1)
  {
    Eigen::Vector3d dur_p = end_path.getPos(i);
    pt.x = dur_p(0);
    pt.y = dur_p(1);
    pt.z = dur_p(2);
    line_strip.points.push_back(pt);
  }

  for (double i = 0; i < P.cols(); i++)
  {
    pt.x = P(0, i);
    pt.y = P(1, i);
    pt.z = P(2, i);
    sphere.points.push_back(pt);
  }

  debug_pub.publish(line_strip);
  debug_pub.publish(sphere);
}

__host__ void TrajOpt::drawDebugWp(std::vector<Eigen::Vector3d> front_path)
{
  int id = 0;
  visualization_msgs::Marker mk;
  mk.header.frame_id = "world";
  mk.header.stamp    = ros::Time::now();
  mk.type            = visualization_msgs::Marker::SPHERE_LIST;
  mk.action          = visualization_msgs::Marker::DELETE;
  mk.id              = id++;
  // kino_pub_.publish(mk);

  mk.action             = visualization_msgs::Marker::ADD;
  mk.pose.orientation.x = 0.0;
  mk.pose.orientation.y = 0.0;
  mk.pose.orientation.z = 0.0;
  mk.pose.orientation.w = 1.0;

  mk.color.r = 0;
  mk.color.g = 1;
  mk.color.b = 0;
  mk.color.a = 1;

  mk.scale.x = 0.075;
  mk.scale.y = 0.075;
  mk.scale.z = 0.075;

  geometry_msgs::Point pt;
  for (int i = 0; i < int(front_path.size()); i++) {
    pt.x = front_path[i](0);
    pt.y = front_path[i](1);
    pt.z = 0;
    mk.points.push_back(pt);
  }
  debug_wp_pub.publish(mk);
  ros::Duration(0.001).sleep();
}
